#include "hip/hip_runtime.h"
#include <sstream>
#include <vector>
#include <chrono>

#include "common.hpp"

__global__ void cuda_empty_kernel() {}

__global__ void cuda_nested_kernel(unsigned int ttl) {
    if (ttl > 0) {
        cuda_nested_kernel<<<1, 1>>>(ttl - 1);
    }
}

void bench_cuda_kernel_latency()
{
    size_t work_sizes[] = {1, 10, 100, 1000};
    cuda_empty_kernel<<<1, 1>>>();
    for(auto ws : work_sizes) {
        std::vector<double> times;
        for (auto i = 0; i < 1000; i ++) {
            auto s = std::chrono::high_resolution_clock::now();
            cuda_empty_kernel<<<ws, 1>>>();
            hipDeviceSynchronize();
            auto e = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration<double, std::micro>(e - s).count());
        }

        std::ostringstream name;
        name << "CUDA kernel launch latency(worksize=" << ws << ")";
        print_result(name.str().c_str(), times);
    }
}

void bench_cuda_nested_kernel_latency()
{
    unsigned int nest_depths[] = {1, 2, 4, 8, 16};
    cuda_nested_kernel<<<1, 1>>>(0);
    for(auto nest_depth : nest_depths) {
        std::vector<double> times;
        for (auto i = 0; i < 10; i ++) {
            auto s = std::chrono::high_resolution_clock::now();
            cuda_nested_kernel<<<1, 1>>>(nest_depth);
            hipDeviceSynchronize();
            auto e = std::chrono::high_resolution_clock::now();
            times.push_back(std::chrono::duration<double, std::micro>(e - s).count());
        }

        std::ostringstream name;
        name << "CUDA nested kernel launch latency(depth=" << nest_depth << ")";
        print_result(name.str().c_str(), times);
    }
}
